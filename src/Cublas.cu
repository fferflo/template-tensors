#include <template_tensors/interface/cublas/Cublas.h>

#ifdef __HIPCC__

namespace cublas {

__host__ __device__
const char* cublasGetErrorString(hipblasStatus_t err)
{
  switch (err)
  {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  return "Unknown CUBLAS error";
}

__host__ __device__
const char* cublasGetOpString(hipblasOperation_t op)
{
  switch (op)
  {
    case HIPBLAS_OP_N: return "HIPBLAS_OP_N";
    case HIPBLAS_OP_T: return "HIPBLAS_OP_T";
    case HIPBLAS_OP_C: return "HIPBLAS_OP_C";
  }
  return "Unknown CUBLAS op";
}

thread_local CublasContext context;

__host__
CublasContext& getContext()
{
  return context;
}

} // end of ns cublas

#endif